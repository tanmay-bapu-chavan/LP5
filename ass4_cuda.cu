#include "hip/hip_runtime.h"
// Program 1
#include <iostream>
#include <vector>
#include <omp.h>
#include <climits>

using namespace std;

void min_reduction(vector<int> &arr)
{
    int min_value = INT_MAX;
#pragma omp parallel for reduction(min : min_value)
    for (int i = 0; i < arr.size(); i++)
    {
        if (arr[i] < min_value)
        {
            min_value = arr[i];
        }
    }
    cout << "Minimum value: " << min_value << endl;
}

void max_reduction(vector<int> &arr)
{
    int max_value = INT_MIN;
#pragma omp parallel for reduction(max : max_value)
    for (int i = 0; i < arr.size(); i++)
    {
        if (arr[i] > max_value)
        {
            max_value = arr[i];
        }
    }
    cout << "Maximum value: " << max_value << endl;
}

void sum_reduction(vector<int> &arr)
{
    int sum = 0;
#pragma omp parallel for reduction(+ : sum)
    for (int i = 0; i < arr.size(); i++)
    {
        sum += arr[i];
    }
    cout << "Sum: " << sum << endl;
}

void average_reduction(vector<int> &arr)
{
    int sum = 0;
#pragma omp parallel for reduction(+ : sum)
    for (int i = 0; i < arr.size(); i++)
    {
        sum += arr[i];
    }
    cout << "Average: " << (double)sum / arr.size() << endl;
}

int main()
{
    vector<int> arr = {5, 2, 9, 1, 7, 6, 8, 3, 4};

    min_reduction(arr);
    max_reduction(arr);
    sum_reduction(arr);
    average_reduction(arr);
}

// Program2

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matmul(int *A, int *B, int *C, int N)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    if (Row < N && Col < N)
    {
        int Pvalue = 0;
        for (int k = 0; k < N; k++)
        {
            Pvalue += A[Row * N + k] * B[k * N + Col];
        }
        C[Row * N + Col] = Pvalue;
    }
}

int main()
{
    int N = 512;
    int size = N * N * sizeof(int);
    int *A, *B, *C;
    int *dev_A, *dev_B, *dev_C;
    hipHostMalloc(&A, size);
    hipHostMalloc(&B, size);
    hipHostMalloc(&C, size);
    hipMalloc(&dev_A, size);
    hipMalloc(&dev_B, size);
    hipMalloc(&dev_C, size);

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = i * N + j;
            B[i * N + j] = j * N + i;
        }
    }

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);

    matmul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, N);

    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++)
    {
        for (int j = 0; j < 10; j++)
        {
            cout << C[i * N + j] << " ";
        }
        cout << endl;
    }

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}


/*

Output:
Program 1 (Reduction Operations):
        Minimum value: 1
        Maximum value: 9
        Sum: 45
        Average: 5

Program 2 (Matrix Multiplication Using CUDA) :
        0 1 2 3 4 5 6 7 8 9 
        1 2 3 4 5 6 7 8 9 10 
        2 3 4 5 6 7 8 9 10 11 
        3 4 5 6 7 8 9 10 11 12 
        4 5 6 7 8 9 10 11 12 13 
        5 6 7 8 9 10 11 12 13 14 
        6 7 8 9 10 11 12 13 14 15 
        7 8 9 10 11 12 13 14 15 16 
        8 9 10 11 12 13 14 15 16 17 
        9 10 11 12 13 14 15 16 17 18 
*/